#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>

#include "common.h"

template <typename T>
class InputGenerator
{
  public:    
    static void Generate(const char *filePath, hipExtent size)
    {
        FILE *file = fopen(filePath, "w+");
        size_t fileSize = size.depth * size.height * size.width * sizeof(T);

        T *data = NULL;  
        MemoryManager::AllocGrayScaleImageCPU<T>(&data, size);
        MatrixAccessor<T> mat(data, size);

        memset(data, 0, fileSize);

        for (size_t z = 0; z < size.depth; z++)
        {
            for (size_t y = 0; y < size.height; y++)
            {
                for (size_t x = 0; x < size.width; x++)
                {
                    if (x % 2 == 0)
                        mat.Get(x, y, z) = x+y+1;
                }
            }
        }

        fwrite(data, fileSize, 1, file);

        fclose(file);
        MemoryManager::FreeCPU(data);
    }
};

typedef struct MaxFilterParams_t
{
    int radius;
} MaxFilterParams;

template <typename val_t>
void MaxFilterCPU(val_t *pInput, val_t *pOutput, hipExtent size, MaxFilterParams params)
{
    MatrixAccessor<val_t> input(pInput, size);
    MatrixAccessor<val_t> output(pOutput, size);

    for (size_t z = 0; z < size.depth; z++)
    for (size_t y = 0; y < size.height; y++)
    for (size_t x = 0; x < size.width; x++)
    {
        val_t maxVal = input.Get(x, y, z);

        for (int oy = -params.radius; oy <= params.radius; oy++)
        for (int ox = -params.radius; ox <= params.radius; ox++)
        {
            if (input.CheckOffsetInBounds(x, y, z, ox, oy, 0))
                maxVal = max(maxVal, input.Get(x+ox, y+oy, z));
        }
        
        output.Get(x, y, z) = maxVal;
    }
}

template <typename val_t>
void MaxFilterParallel(val_t *pInput, val_t *pOutput, hipExtent size, MaxFilterParams params, bool gpu)
{
    auto kernel = [=]__host__ __device__(dim3 blockIdx, dim3 blockDim, dim3 threadIdx)
    {
        MatrixAccessor<val_t> input(pInput, size);
        MatrixAccessor<val_t> output(pOutput, size);

        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;
        size_t z = blockIdx.z * blockDim.z + threadIdx.z;
        
        if (x < size.width && y < size.height && z < size.depth)
        {
            val_t maxVal = input.Get(x, y, z);

            for (int oy = -params.radius; oy <= params.radius; oy++)
            for (int ox = -params.radius; ox <= params.radius; ox++)
            {
                if (input.CheckOffsetInBounds(x, y, z, ox, oy, 0))
                    maxVal = max(maxVal, input.Get(x+ox, y+oy, z));
            }            

            output.Get(x, y, z) = maxVal;
        }
    };

    if (gpu)
    {
        const size_t BLOCK_SIZE = 8;

        dim3 blocks(DIVUP(size.width, BLOCK_SIZE), DIVUP(size.height, BLOCK_SIZE), DIVUP(size.depth, BLOCK_SIZE));
        dim3 threads(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);

        lambda_invoker<<<blocks, threads>>>(kernel);

        auto e = hipDeviceSynchronize();
        CHECK_CUDA_ERROR(e);
    }
    else
    {        
        const size_t BLOCK_SIZE = 512;

        dim3 blocks(DIVUP(size.width, BLOCK_SIZE), DIVUP(size.height, BLOCK_SIZE), DIVUP(size.depth, BLOCK_SIZE));

        for (size_t bz = 0; bz < blocks.z; bz++)
        for (size_t by = 0; by < blocks.y; by++)
        for (size_t bx = 0; bx < blocks.x; bx++)
        {
            dim3 blockIdx(bx, by, bz);

            for (size_t tz = 0; tz < BLOCK_SIZE; tz++)
            for (size_t ty = 0; ty < BLOCK_SIZE; ty++)
            for (size_t tx = 0; tx < BLOCK_SIZE; tx++)
            {
                dim3 threadIdx(tx, ty, tz);
                dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
                kernel(blockIdx, blockDim, threadIdx);
            }
        }
    }    
}

int main(int argc, const char *argv[])
{
    //nvcc main.cu --expt-extended-lambda --std=c++11 -Xcompiler -fopenmp --compiler-options -fPIC -O3 -D__INTEL_COMPILER -o main

    InputGenerator<float>::Generate("test.b", make_hipExtent(10, 10, 10));

    const char *inputPath = argv[1];
    const char *outputPath = argv[2];
    size_t width = atoi(argv[3]);
    size_t height = atoi(argv[4]);
    size_t depth = atoi(argv[5]);

    hipExtent size = make_hipExtent(width, height, depth);

    float *input = NULL;
    float *output = NULL;
    MemoryManager::AllocGrayScaleImageCPU<float>(&input, size);
    MemoryManager::AllocGrayScaleImageCPU<float>(&output, size);
    
    FileManager::ReadAs<unsigned short, float>(input, inputPath, size);

    bool gpu = true;

    CudaTimer timer;
    printf("MaxFilter\n");
    MaxFilterParams params;
    params.radius = 8;

    timer.start();

    if (gpu)
    {
        float *d_input = NULL;
        float *d_output = NULL;
        MemoryManager::AllocGrayScaleImageGPU<float>(&d_input, size);
        MemoryManager::AllocGrayScaleImageGPU<float>(&d_output, size);
        MemoryManager::CopyGrayScaleImageToGPU<float>(input, d_input, size);

        MaxFilterParallel<float>(d_input, d_output, size, params, true);

        MemoryManager::CopyGrayScaleImageFromGPU<float>(output, d_output, size);
        MemoryManager::FreeGPU(d_input);
        MemoryManager::FreeGPU(d_output);
    }
    else
    {
        MaxFilterParallel<float>(input, output, size, params, false);        
    }
    
    timer.stop();

    printf("MaxFilter %fms\n", timer.elapsedMs);

    FileManager::Write<float>(output, outputPath, size);

    MemoryManager::FreeCPU(input);
    MemoryManager::FreeCPU(output);    
}
