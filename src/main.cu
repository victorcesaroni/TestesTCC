#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>

#include "common.h"

template <typename T>
class InputGenerator
{
  public:    
    static void Generate(const char *filePath, hipExtent size)
    {
        FILE *file = fopen(filePath, "w+");
        size_t fileSize = size.depth * size.height * size.width * sizeof(T);

        T *data = NULL;  
        MemoryManager::AllocGrayScaleImageCPU<T>(&data, size);
        MatrixAccessor<T> mat(data, size);

        memset(data, 0, fileSize);

        for (size_t z = 0; z < size.depth; z++)
        {
            for (size_t y = 0; y < size.height; y++)
            {
                for (size_t x = 0; x < size.width; x++)
                {
                    if (x % 2 == 0)
                        mat.Get(x, y, z) = x+y+1;
                }
            }
        }

        fwrite(data, fileSize, 1, file);

        fclose(file);
        MemoryManager::FreeCPU(data);
    }
};

typedef struct MaxFilterParams_t
{
    int radius;
} MaxFilterParams;

template <typename val_t>
void MaxFilterCPU(val_t *pInput, val_t *pOutput, hipExtent size, MaxFilterParams params)
{
    MatrixAccessor<val_t> input(pInput, size);
    MatrixAccessor<val_t> output(pOutput, size);

    for (size_t z = 0; z < size.depth; z++)
    for (size_t y = 0; y < size.height; y++)
    for (size_t x = 0; x < size.width; x++)
    {
        val_t maxVal = input.Get(x, y, z);

        for (int oy = -params.radius; oy <= params.radius; oy++)
        for (int ox = -params.radius; ox <= params.radius; ox++)
        {
            if (input.CheckOffsetInBounds(x, y, z, ox, oy, 0))
                maxVal = max(maxVal, input.Get(x+ox, y+oy, z));
        }
        
        output.Get(x, y, z) = maxVal;
    }
}


template <typename val_t>
__global__ void CopyKernel(val_t *d_input, val_t *d_output, hipExtent size)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t z = blockIdx.z * blockDim.z + threadIdx.z;

    MatrixAccessor<val_t> input(d_input, size);
    MatrixAccessor<val_t> output(d_output, size);
    
    if (x < size.width && y < size.height && z < size.depth)
    {
        output.Get(x, y, z) = input.Get(x, y, z);
    }
}

void TestCUDA()
{
    printf("TEST CUDA\n");

    hipExtent size = make_hipExtent(4096, 1023, 17);

    size_t bytes = size.depth * size.height * size.width * sizeof(float);
    
    printf("Alloc CPU\n");
    float *pInput = (float*)malloc(bytes);
    float *pOutput = (float*)malloc(bytes);

    float *d_input;
    float *d_output;
    hipError_t e;    

    printf("Alloc GPU\n");
    e = hipMalloc(&d_input, bytes);
    CHECK_CUDA_ERROR(e);
    e = hipMalloc(&d_output, bytes);
    CHECK_CUDA_ERROR(e);

    MatrixAccessor<float> input(pInput, size);
    MatrixAccessor<float> output(pOutput, size);

    for (size_t z = 0; z < size.depth; z++)
    for (size_t y = 0; y < size.height; y++)
    for (size_t x = 0; x < size.width; x++)
    {
        input.Get(x,y,z) = (float)(rand() % 255);
    }

    printf("H2D\n");
    e = hipMemcpy(d_input, pInput, bytes, hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(e);
    
    printf("KERNEL\n");
    const size_t BLOCK_SIZE = 8;
    dim3 blocks(DIVUP(size.width, BLOCK_SIZE), DIVUP(size.height, BLOCK_SIZE), DIVUP(size.depth, BLOCK_SIZE));
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);

    CopyKernel<<<blocks,threads>>>(d_input, d_output, size);
    hipDeviceSynchronize();

    printf("D2H\n");
    e = hipMemcpy(pOutput, d_output, bytes, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(e);

    for (size_t z = 0; z < size.depth; z++)
    for (size_t y = 0; y < size.height; y++)
    {
        for (size_t x = 0; x < size.width; x++)
        {
            float a = input.Get(x,y,z);
            float b = output.Get(x,y,z);

            //printf("%f ", b);
            if (fabsf(fabsf(a) - fabsf(b)) > 0.01f)
            {
                printf("TEST CUDA ERROR %f %f\n", a, b);
            }
        }
        //printf("\n");
    }

    printf("TEST END\n");

    free(pInput);
    free(pOutput);

    hipFree(d_input);
    hipFree(d_output);
}

template <typename val_t>
void MaxFilterParallel(val_t *pInput, val_t *pOutput, hipExtent size, MaxFilterParams params, bool gpu)
{
    auto kernel = [pInput, pOutput, size, params] __host__ __device__ (dim3 blockIdx, dim3 blockDim, dim3 threadIdx)
    {
        MatrixAccessor<val_t> input(pInput, size);
        MatrixAccessor<val_t> output(pOutput, size);

        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;
        size_t z = blockIdx.z * blockDim.z + threadIdx.z;
        
        if (x < size.width && y < size.height && z < size.depth)
        {
            val_t maxVal = input.Get(x, y, z);

            for (int oy = -params.radius; oy <= params.radius; oy++)
            for (int ox = -params.radius; ox <= params.radius; ox++)
            {
                if (input.CheckOffsetInBounds(x, y, z, ox, oy, 0))
                    maxVal = max(maxVal, input.Get(x+ox, y+oy, z));
            }            

            output.Get(x, y, z) = maxVal;
        }
    };

    if (gpu)
    {
        const size_t BLOCK_SIZE = 8;

        dim3 blocks(DIVUP(size.width, BLOCK_SIZE), DIVUP(size.height, BLOCK_SIZE), DIVUP(size.depth, BLOCK_SIZE));
        dim3 threads(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);

        lambda_invoker<<<blocks, threads>>>(kernel);

        auto e = hipDeviceSynchronize();
        CHECK_CUDA_ERROR(e);
    }
    else
    {        
        const size_t BLOCK_SIZE = 512;

        dim3 blocks(DIVUP(size.width, BLOCK_SIZE), DIVUP(size.height, BLOCK_SIZE), DIVUP(size.depth, BLOCK_SIZE));
        dim3 threads(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);

        for (size_t bz = 0; bz < blocks.z; bz++)
        for (size_t by = 0; by < blocks.y; by++)
        for (size_t bx = 0; bx < blocks.x; bx++)
        {
            dim3 blockIdx(bx, by, bz);

            for (size_t tz = 0; tz < threads.z; tz++)
            for (size_t ty = 0; ty < threads.y; ty++)
            for (size_t tx = 0; tx < threads.x; tx++)
            {
                dim3 threadIdx(tx, ty, tz);
                dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
                kernel(blockIdx, blockDim, threadIdx);
            }
        }
    }    
}

int main(int argc, const char *argv[])
{
    //nvcc main.cu --expt-extended-lambda --std=c++11 -Xcompiler -fopenmp --compiler-options -fPIC -O3 -D__INTEL_COMPILER -o main
    //nvcc main.cu -gencode arch=compute_61,code=[sm_61,compute_61] --expt-extended-lambda --std=c++11 -Xcompiler -fopenmp --compiler-options -fPIC -O3 -D__INTEL_COMPILER -DCMAKE_C_COMPILER=/usr/bin/gcc-6 -o main
    
    TestCUDA();

    InputGenerator<float>::Generate("test.b", make_hipExtent(10, 10, 1));

    const char *inputPath = argv[1];
    const char *outputPath = argv[2];
    size_t width = atoi(argv[3]);
    size_t height = atoi(argv[4]);
    size_t depth = atoi(argv[5]);

    hipExtent size = make_hipExtent(width, height, depth);

    float *input = NULL;
    float *output = NULL;
    MemoryManager::AllocGrayScaleImageCPU<float>(&input, size);
    MemoryManager::AllocGrayScaleImageCPU<float>(&output, size);
    
    FileManager::ReadAs<unsigned short, float>(input, inputPath, size);

    bool gpu = true;

    CudaTimer timer;
    printf("MaxFilter\n");
    MaxFilterParams params;
    params.radius = 8;

    timer.start();

    if (gpu)
    {
        float *d_input = NULL;
        float *d_output = NULL;
        MemoryManager::AllocGrayScaleImageGPU<float>(&d_input, size);
        MemoryManager::AllocGrayScaleImageGPU<float>(&d_output, size);
        MemoryManager::CopyGrayScaleImageToGPU<float>(input, d_input, size);

        MaxFilterParallel<float>(d_input, d_output, size, params, true);

        MemoryManager::CopyGrayScaleImageFromGPU<float>(output, d_output, size);
        MemoryManager::FreeGPU(d_input);
        MemoryManager::FreeGPU(d_output);
    }
    else
    {
        MaxFilterParallel<float>(input, output, size, params, false);        
    }
    
    timer.stop();

    printf("MaxFilter %fms\n", timer.elapsedMs);

    FileManager::Write<float>(output, outputPath, size);

    MemoryManager::FreeCPU(input);
    MemoryManager::FreeCPU(output);    
}
