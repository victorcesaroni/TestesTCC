#include "cuda_timer.h"

CudaTimer::CudaTimer()
{
    hipEventCreate(&starte);
    hipEventCreate(&stope);
}

CudaTimer::~CudaTimer()
{
    hipEventDestroy(starte);
    hipEventDestroy(stope);
}

void CudaTimer::start()
{
    hipEventRecord(starte);
    
    elapsedMs=0;
}

float CudaTimer::stop()
{
    hipEventRecord(stope);
    hipEventSynchronize(stope);
    hipEventElapsedTime(&elapsedMs, starte, stope);

    return elapsedMs;
}
